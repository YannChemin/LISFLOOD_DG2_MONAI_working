#include "hip/hip_runtime.h"
#include "get_level.cuh"

__device__ int lis::cuda::acc_nugrid::get_level(index_1D idx)
{
	return log( C(3.0) * idx + 1) / log( C(4.0) );
}