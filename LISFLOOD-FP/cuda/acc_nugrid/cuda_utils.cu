#include "cuda_utils.cuh"

hipError_t lis::cuda::acc_nugrid::sync()
{
	return hipDeviceSynchronize();
}

hipError_t lis::cuda::acc_nugrid::peek()
{
	return hipPeekAtLastError();
}

hipError_t lis::cuda::acc_nugrid::reset()
{
	return hipDeviceReset();
}

hipError_t lis::cuda::acc_nugrid::copy_cuda
(
	void* dst,
	void* src,
	size_t bytes
)
{
	hipError_t error = hipMemcpy
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	);

	return error;
}

__host__ __device__
void* lis::cuda::acc_nugrid::malloc_device
(
	size_t bytes
)
{
	void* ptr;
	
	hipMalloc
	(
		&ptr,
		bytes
	);

	return ptr;
}

__host__ __device__
void* lis::cuda::acc_nugrid::malloc_unified
(
	size_t bytes
)
{
	void* ptr;

	hipMallocManaged
	(
		&ptr,
		bytes
	);

	return ptr;
}


__host__ __device__
hipError_t lis::cuda::acc_nugrid::free_device
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipFree(ptr) : hipSuccess;
}

__host__ __device__
hipError_t lis::cuda::acc_nugrid::free_unified
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipFree(ptr) : hipSuccess;
}